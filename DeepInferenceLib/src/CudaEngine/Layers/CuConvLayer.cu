#include "CudaEngine/Layers/CuConvLayer.cuh"

hipdnnHandle_t CuConvLayer::handle;

/// @brief Use this for Creating Conv input-layer 
/// @param inputChannelCount 
/// @param outputChannelCount 
/// @param widthOfChannels 
/// @param heightOfChannels 
/// @param padding 
/// @param stride 
/// @param dilation 
/// @param inputImageBatchSize 
/// @param inputImageChannels 
/// @param inputImageHeight 
/// @param inputImageWidth 
/// @param activation 
CuConvLayer::CuConvLayer(int inputChannelCount, int outputChannelCount, int widthOfChannels, int heightOfChannels, 
    int padding, int stride, int dilation,
    int inputImageBatchSize, int inputImageChannels, int inputImageHeight, int inputImageWidth, Activation activation)
{
    this->isInputLayer = true;
    checkCUDNN(hipdnnCreateTensorDescriptor(this->cuInput.descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(*this->cuInput.descriptor,
        /*format=*/HIPDNN_TENSOR_NHWC,
        /*dataType=*/HIPDNN_DATA_FLOAT,
        /*batch_size=*/inputImageBatchSize,
        /*channels=*/inputImageChannels,
        /*image_height=*/inputImageHeight,
        /*image_width=*/inputImageWidth));
    this->cuInput.sizeInBytes = sizeof(float) * inputImageBatchSize * inputImageChannels * inputImageHeight * inputImageWidth;
    hipMalloc(&(this->cuInput.onDevice), this->cuInput.sizeInBytes);

    checkCUDNN(hipdnnCreateFilterDescriptor(this->cuKernel.descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(*this->cuKernel.descriptor,
        /*dataType=*/HIPDNN_DATA_FLOAT,
        /*format=*/HIPDNN_TENSOR_NCHW,
        /*out_channels=*/outputChannelCount,
        /*in_channels=*/inputChannelCount,
        /*kernel_height=*/heightOfChannels,
        /*kernel_width=*/widthOfChannels));

    checkCUDNN(hipdnnCreateConvolutionDescriptor(this->cuConvolution.descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(*this->cuConvolution.descriptor,
        /*pad_height=*/padding,
        /*pad_width=*/padding,
        /*vertical_stride=*/stride,
        /*horizontal_stride=*/stride,
        /*dilation_height=*/dilation,
        /*dilation_width=*/dilation,
        /*mode=*/HIPDNN_CROSS_CORRELATION,
        /*computeType=*/HIPDNN_DATA_FLOAT));
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(*this->cuConvolution.descriptor,
        *this->cuInput.descriptor,
        *this->cuKernel.descriptor,
        &(this->cuOutput.batchSize),
        &(this->cuOutput.channelCount),
        &(this->cuOutput.height),
        &(this->cuOutput.width)));

    Tensor4D& cuOut = this->cuOutput;
    checkCUDNN(hipdnnCreateTensorDescriptor(cuOut.descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(*cuOut.descriptor,
        /*format=*/HIPDNN_TENSOR_NHWC,
        /*dataType=*/HIPDNN_DATA_FLOAT,
        /*batch_size=*/cuOut.batchSize,
        /*channels=*/cuOut.channelCount,
        /*image_height=*/cuOut.height,
        /*image_width=*/cuOut.width));
    cuOut.sizeInBytes = sizeof(float) * cuOut.batchSize * cuOut.channelCount * cuOut.height * cuOut.width;
    hipMalloc(&(cuOut.onDevice), cuOut.sizeInBytes);
    hipMemset(cuOut.onDevice, 0, cuOut.sizeInBytes);

    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(CuConvLayer::handle,
        *this->cuInput.descriptor,
        *this->cuKernel.descriptor,
        *this->cuConvolution.descriptor,
        *this->cuOutput.descriptor,
        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
        /*memoryLimitInBytes=*/0,
        this->cuConvolution.algo));

    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(CuConvLayer::handle,
        *this->cuInput.descriptor,
        *this->cuKernel.descriptor,
        *this->cuConvolution.descriptor,
        *this->cuOutput.descriptor,
        *this->cuConvolution.algo,
        &(this->cuWorkspace.sizeInBytes)));
    hipMalloc(&this->cuWorkspace.onDevice, this->cuWorkspace.sizeInBytes);

    this->activation = activation;
}

/// @brief Use if for creating hidden Conv input-layers
/// @param inputChannelCount 
/// @param outputChannelCount 
/// @param widthOfChannels 
/// @param heightOfChannels 
/// @param padding 
/// @param stride 
/// @param dilation 
/// @param prevLayer 
/// @param activation 
CuConvLayer::CuConvLayer(int inputChannelCount, int outputChannelCount, int widthOfChannels, int heightOfChannels,
    int padding, int stride, int dilation, 
    const CuConvLayer* prevLayer, Activation activation)
{
    this->cuInput = std::move(prevLayer->cuOutput);

    checkCUDNN(hipdnnCreateFilterDescriptor(this->cuKernel.descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(*this->cuKernel.descriptor,
        /*dataType=*/HIPDNN_DATA_FLOAT,
        /*format=*/HIPDNN_TENSOR_NCHW,
        /*out_channels=*/outputChannelCount,
        /*in_channels=*/inputChannelCount,
        /*kernel_height=*/heightOfChannels,
        /*kernel_width=*/widthOfChannels));
    checkCUDA(hipMalloc(&(this->cuKernel.onDevice), this->cuKernel.sizeInBytes));

    checkCUDNN(hipdnnCreateConvolutionDescriptor(this->cuConvolution.descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(*this->cuConvolution.descriptor,
        /*pad_height=*/padding,
        /*pad_width=*/padding,
        /*vertical_stride=*/stride,
        /*horizontal_stride=*/stride,
        /*dilation_height=*/dilation,
        /*dilation_width=*/dilation,
        /*mode=*/HIPDNN_CROSS_CORRELATION,
        /*computeType=*/HIPDNN_DATA_FLOAT));
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(*this->cuConvolution.descriptor,
        *this->cuInput.descriptor,
        *this->cuKernel.descriptor,
        &(this->cuOutput.batchSize),
        &(this->cuOutput.channelCount),
        &(this->cuOutput.height),
        &(this->cuOutput.width)));

    Tensor4D& cuOut = this->cuOutput;
    checkCUDNN(hipdnnCreateTensorDescriptor(cuOut.descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(*cuOut.descriptor,
        /*format=*/HIPDNN_TENSOR_NHWC,
        /*dataType=*/HIPDNN_DATA_FLOAT,
        /*batch_size=*/cuOut.batchSize,
        /*channels=*/cuOut.channelCount,
        /*image_height=*/cuOut.height,
        /*image_width=*/cuOut.width));
    cuOut.sizeInBytes = sizeof(float) * cuOut.batchSize * cuOut.channelCount * cuOut.height * cuOut.width;
    checkCUDA(hipMalloc(&(cuOut.onDevice), cuOut.sizeInBytes));
    checkCUDA(hipMemset(cuOut.onDevice, 0, cuOut.sizeInBytes));

    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(CuConvLayer::handle,
        *this->cuInput.descriptor,
        *this->cuKernel.descriptor,
        *this->cuConvolution.descriptor,
        *this->cuOutput.descriptor,
        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
        /*memoryLimitInBytes=*/0,
        this->cuConvolution.algo));

    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(CuConvLayer::handle,
        *this->cuInput.descriptor,
        *this->cuKernel.descriptor,
        *this->cuConvolution.descriptor,
        *this->cuOutput.descriptor,
        *this->cuConvolution.algo,
        &(this->cuWorkspace.sizeInBytes)));
    checkCUDA(hipMalloc(&this->cuWorkspace.onDevice, this->cuWorkspace.sizeInBytes));

    this->activation = activation;
}

CuConvLayer::~CuConvLayer()
{
}

/// @brief 
/// @param weights are in NCHW order
/// @param numberOfWeights 
/// @param bias it's not supported
/// @param numberOfBias it's not supported
void CuConvLayer::init(const float* weights, const int numberOfWeights, const float* bias, const int numberOfBias)
{
    CuKernel& cuKernel = this->cuKernel;
    if (numberOfWeights * sizeof(float) != cuKernel.sizeInBytes)
        throw "Number of weights received doesn't fit the expectation.";
    checkCUDA(hipMemcpy(cuKernel.onDevice, weights, cuKernel.sizeInBytes, hipMemcpyHostToDevice));
    hipDeviceSynchronize();
}

float* CuConvLayer::compute(const float* x)
{
    const float alpha = 1.0f, beta = 0.0f;

    checkCUDNN(hipdnnConvolutionForward(CuConvLayer::handle,
        &alpha,
        *this->cuInput.descriptor,
        this->cuInput.dataOnDevice,
        *this->cuKernel.descriptor,
        this->cuKernel.onDevice,
        *this->cuConvolution.descriptor,
        *this->cuConvolution.algo,
        this->cuWorkspace.onDevice,
        this->cuWorkspace.sizeInBytes,
        &beta,
        *this->cuOutput.descriptor,
        this->cuOutput.onDevice));
    hipDeviceSynchronize();
}

const Tensor4D& CuConvLayer::getOutputOnDevice() const
{
    return this->cuOutput;
}

std::vector<float>&& CuConvLayer::getOutput() const
{
    const Tensor4D& output = this->cuOutput;
    return std::vector<float>(output.onDevice, output.onDevice + output.sizeInBytes / sizeof(float));
}

