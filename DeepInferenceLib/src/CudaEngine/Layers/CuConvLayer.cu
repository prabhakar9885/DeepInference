#include "CudaEngine/Layers/CuConvLayer.cuh"

hipdnnHandle_t CuConvLayer::handle;

void CuConvLayer::allocMemForLayer()
{
    if (this->hasInputLayer()) {
        checkCUDNN(hipdnnCreateTensorDescriptor(&this->cuInput.descriptor));
        checkCUDNN(hipdnnSetTensor4dDescriptor(this->cuInput.descriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/this->cuInput.batchSize,
            /*channels=*/this->cuInput.channelCount,
            /*image_height=*/this->cuInput.height,
            /*image_width=*/this->cuInput.width));
        hipMallocManaged(&(this->cuInput.onDevice), this->cuInput.sizeInBytes);
    }

    checkCUDNN(hipdnnCreateFilterDescriptor(&this->cuKernel.weightDescriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(this->cuKernel.weightDescriptor,
        /*dataType=*/HIPDNN_DATA_FLOAT,
        /*format=*/HIPDNN_TENSOR_NCHW,
        /*out_channels=*/ this->cuKernel.outChannels,
        /*in_channels=*/ this->cuKernel.inChannels,
        /*kernel_height=*/ this->cuKernel.heightOfChannel,
        /*kernel_width=*/ this->cuKernel.widthOfChannel));
    hipMallocManaged(&(this->cuKernel.weightOnDevice), this->cuKernel.sizeofWeightsInBytes);

    checkCUDNN(hipdnnCreateTensorDescriptor(&this->cuKernel.biasDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(this->cuKernel.biasDescriptor,
        /*format=*/HIPDNN_TENSOR_NHWC,
        /*dataType=*/HIPDNN_DATA_FLOAT,
        /*batch_size=*/this->cuKernel.outChannels,
        /*channels=*/ this->cuKernel.inChannels,
        /*image_height=*/ this->cuKernel.heightOfChannel,
        /*image_width=*/ this->cuKernel.widthOfChannel));
    hipMallocManaged(&(this->cuKernel.biasOnDevice), this->cuKernel.sizeofBiasInBytes);

    checkCUDNN(hipdnnCreateConvolutionDescriptor(&this->cuConvolution.descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(this->cuConvolution.descriptor,
        /*pad_height=*/this->cuConvolution.padding,
        /*pad_width=*/this->cuConvolution.padding,
        /*vertical_stride=*/this->cuConvolution.stride,
        /*horizontal_stride=*/this->cuConvolution.stride,
        /*dilation_height=*/this->cuConvolution.dilation,
        /*dilation_width=*/this->cuConvolution.dilation,
        /*mode=*/HIPDNN_CROSS_CORRELATION,
        /*computeType=*/HIPDNN_DATA_FLOAT));
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(this->cuConvolution.descriptor,
        this->cuInput.descriptor,
        this->cuKernel.weightDescriptor,
        &(this->cuOutput.batchSize),
        &(this->cuOutput.channelCount),
        &(this->cuOutput.height),
        &(this->cuOutput.width)));

    Tensor4D& cuOut = this->cuOutput;
    checkCUDNN(hipdnnCreateTensorDescriptor(&cuOut.descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(cuOut.descriptor,
        /*format=*/HIPDNN_TENSOR_NHWC,
        /*dataType=*/HIPDNN_DATA_FLOAT,
        /*batch_size=*/cuOut.batchSize,
        /*channels=*/cuOut.channelCount,
        /*image_height=*/cuOut.height,
        /*image_width=*/cuOut.width));
    cuOut.sizeInBytes = sizeof(float) * cuOut.batchSize * cuOut.channelCount * cuOut.height * cuOut.width;
    hipMallocManaged(&(cuOut.onDevice), cuOut.sizeInBytes);
    hipMemset(cuOut.onDevice, 0, cuOut.sizeInBytes);

    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(CuConvLayer::handle,
        this->cuInput.descriptor,
        this->cuKernel.weightDescriptor,
        this->cuConvolution.descriptor,
        this->cuOutput.descriptor,
        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
        /*memoryLimitInBytes=*/0,
        &this->cuConvolution.algo));

    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(CuConvLayer::handle,
        this->cuInput.descriptor,
        this->cuKernel.weightDescriptor,
        this->cuConvolution.descriptor,
        this->cuOutput.descriptor,
        this->cuConvolution.algo,
        &(this->cuWorkspace.sizeInBytes)));
    hipMallocManaged(&this->cuWorkspace.onDevice, this->cuWorkspace.sizeInBytes);
}


CuConvLayer::CuConvLayer(int inputChannelCount, int outputChannelCount, int widthOfChannels, int heightOfChannels,
    int padding, int stride, int dilation, Activation activation)
{
    this->cuKernel.outChannels = outputChannelCount;
    this->cuKernel.inChannels = inputChannelCount;
    this->cuKernel.heightOfChannel = heightOfChannels;
    this->cuKernel.widthOfChannel = widthOfChannels;
    this->cuKernel.sizeofWeightsInBytes = sizeof(float) * outputChannelCount * inputChannelCount * heightOfChannels * widthOfChannels;
    this->cuKernel.sizeofBiasInBytes = sizeof(float) * outputChannelCount;

    this->cuConvolution.padding = padding;
    this->cuConvolution.stride = stride;
    this->cuConvolution.dilation = dilation;

    this->activation = activation;
}

/// @brief Use this for Creating Conv input-layer 
/// @param inputChannelCount 
/// @param outputChannelCount 
/// @param widthOfChannels 
/// @param heightOfChannels 
/// @param padding 
/// @param stride 
/// @param dilation 
/// @param inputImageBatchSize 
/// @param inputImageChannels 
/// @param inputImageHeight 
/// @param inputImageWidth 
/// @param activation 
CuConvLayer::CuConvLayer(int inputChannelCount, int outputChannelCount, int heightOfChannels, int widthOfChannels, 
    int padding, int stride, int dilation,
    int inputImageBatchSize, int inputImageChannels, int inputImageHeight, int inputImageWidth, Activation activation) : 
        CuConvLayer(inputChannelCount, outputChannelCount, widthOfChannels, heightOfChannels, padding, stride, dilation, activation)
{
    this->isInputLayer = true;
    hipSetDevice(0);
    checkCUDNN(hipdnnCreate(&CuConvLayer::handle));

    int versionUsed, maxVersion;
    hipRuntimeGetVersion(&versionUsed);
    hipDriverGetVersion(&maxVersion);
    std::cout << "CUDA version: " << versionUsed << "/" << maxVersion << "\n";
    std::cout << "cuDNN version: " << hipdnnGetVersion() << "\n";

    this->cuInput.batchSize = inputImageBatchSize;
    this->cuInput.channelCount = inputImageChannels;
    this->cuInput.height = inputImageHeight;
    this->cuInput.width = inputImageWidth;
    this->cuInput.sizeInBytes = sizeof(float) * inputImageBatchSize * inputImageChannels * inputImageHeight * inputImageWidth;
}

/// @brief Use if for creating hidden Conv input-layers
/// @param inputChannelCount 
/// @param outputChannelCount 
/// @param widthOfChannels 
/// @param heightOfChannels 
/// @param padding 
/// @param stride 
/// @param dilation 
/// @param prevLayer 
/// @param activation 
CuConvLayer::CuConvLayer(int inputChannelCount, int outputChannelCount, int widthOfChannels, int heightOfChannels,
    int padding, int stride, int dilation, 
    const CuConvLayer* prevLayer, Activation activation) :
        CuConvLayer(inputChannelCount, outputChannelCount, widthOfChannels, heightOfChannels, padding, stride, dilation, activation)
{
    this->cuInput = std::move(prevLayer->cuOutput);
}

CuConvLayer::~CuConvLayer()
{
}

/// @brief 
/// @param weights are in NCHW order
/// @param numberOfWeights 
/// @param bias it's not supported
/// @param numberOfBias it's not supported
void CuConvLayer::init(const float* weights, const int numberOfWeights, const float* bias, const int numberOfBias)
{
    CuKernel& cuKernel = this->cuKernel;
    if (numberOfWeights * sizeof(float) != cuKernel.sizeofWeightsInBytes)
        throw "Number of weights received doesn't fit the expectation.";
    if (numberOfBias * sizeof(float) != cuKernel.sizeofBiasInBytes)
        throw "Number of Biases received doesn't fit the expectation.";
    checkCUDA(hipMemcpy(cuKernel.weightOnDevice, weights, cuKernel.sizeofWeightsInBytes, hipMemcpyHostToDevice));
    checkCUDA(hipMemcpy(cuKernel.biasOnDevice, bias, cuKernel.sizeofBiasInBytes, hipMemcpyHostToDevice));
    hipDeviceSynchronize();
}

float* CuConvLayer::compute(const float* x)
{
    const float alpha = 1.0f, beta = 0.0f;

    if (this->hasInputLayer())
        checkCUDA(hipMemcpy(cuInput.onDevice, x, cuInput.sizeInBytes, hipMemcpyHostToDevice));

    checkCUDNN(hipdnnConvolutionForward(CuConvLayer::handle,
        &alpha,
        this->cuInput.descriptor,
        this->cuInput.onDevice,
        this->cuKernel.weightDescriptor,
        this->cuKernel.weightOnDevice,
        this->cuConvolution.descriptor,
        this->cuConvolution.algo,
        this->cuWorkspace.onDevice,
        this->cuWorkspace.sizeInBytes,
        &beta,
        this->cuOutput.descriptor,
        this->cuOutput.onDevice));
    hipDeviceSynchronize();

    CuUtills::addBiasForNHWC(
        this->cuOutput.onDevice,
        this->cuKernel.biasOnDevice,
        this->cuOutput.batchSize,
        this->cuOutput.height,
        this->cuOutput.width,
        this->cuOutput.channelCount
        );
    hipDeviceSynchronize();

    CuUtills::computeActivation(
        this->cuOutput.onDevice,
        this->cuOutput.sizeInBytes / sizeof(float),
        Activation::NONE);
    hipDeviceSynchronize();

    #ifdef DEBUG
        std::cout << "\n=x=x=x=x=x=x=x=x=x=x=x=x=x=x=";
        int N = cuOutput.batchSize;
        int C = cuOutput.channelCount;
        int H = cuOutput.height;
        int W = cuOutput.width;
        float* data = new float[(long long)N * C * H * W];
        checkCUDA(hipMemcpy(data, cuOutput.onDevice, cuOutput.sizeInBytes, hipMemcpyDeviceToHost));
        int i = 0;
        for (int n = 0; n < N; n++)
        {
            std::cout << "\nBatch: " << N;
            for (int c = 0; c < C; c++)
            {
                std::cout << "\nChannel: " << c << " => \n";
                for (int h = 0; h < H; h++)
                {
                    for (int w = 0; w < W; w++)
                        std::cout << std::setw(6) << data[c + (w + (h + (n)*H) * W) * C] << "/" << data[i++];
                    std::cout << "\n";
                }
            }
        }
        delete(data);
    #endif // DEBUG
        return cuOutput.onDevice;
}

const Tensor4D& CuConvLayer::getOutputOnDevice() const
{
    return this->cuOutput;
}

std::vector<float>&& CuConvLayer::getOutput() const
{
    const Tensor4D& output = this->cuOutput;
    return std::vector<float>(output.onDevice, output.onDevice + output.sizeInBytes / sizeof(float));
}

