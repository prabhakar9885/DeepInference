#include "CudaEngine/Layers/CuConvLayer.cuh"

hipdnnHandle_t CuConvLayer::handle;

/// @brief Use this for Creating Conv input-layer 
/// @param inputChannelCount 
/// @param outputChannelCount 
/// @param widthOfChannels 
/// @param heightOfChannels 
/// @param padding 
/// @param stride 
/// @param dilation 
/// @param inputImageBatchSize 
/// @param inputImageChannels 
/// @param inputImageHeight 
/// @param inputImageWidth 
/// @param activation 
CuConvLayer::CuConvLayer(int inputChannelCount, int outputChannelCount, int heightOfChannels, int widthOfChannels, 
    int padding, int stride, int dilation,
    int inputImageBatchSize, int inputImageChannels, int inputImageHeight, int inputImageWidth, Activation activation)
{
    this->isInputLayer = true;
    hipSetDevice(0);
    checkCUDNN(hipdnnCreate(&CuConvLayer::handle));

    checkCUDNN(hipdnnCreateTensorDescriptor(&this->cuInput.descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(this->cuInput.descriptor,
        /*format=*/HIPDNN_TENSOR_NHWC,
        /*dataType=*/HIPDNN_DATA_FLOAT,
        /*batch_size=*/inputImageBatchSize,
        /*channels=*/inputImageChannels,
        /*image_height=*/inputImageHeight,
        /*image_width=*/inputImageWidth));
    this->cuInput.sizeInBytes = sizeof(float) * inputImageBatchSize * inputImageChannels * inputImageHeight * inputImageWidth;
    hipMallocManaged(&(this->cuInput.onDevice), this->cuInput.sizeInBytes);

    checkCUDNN(hipdnnCreateFilterDescriptor(&this->cuKernel.descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(this->cuKernel.descriptor,
        /*dataType=*/HIPDNN_DATA_FLOAT,
        /*format=*/HIPDNN_TENSOR_NCHW,
        /*out_channels=*/ (this->cuKernel.outChannels = outputChannelCount),
        /*in_channels=*/ (this->cuKernel.inChannels = inputChannelCount),
        /*kernel_height=*/ (this->cuKernel.heightOfChannel = heightOfChannels),
        /*kernel_width=*/ (this->cuKernel.widthOfChannel = widthOfChannels)));
    this->cuKernel.sizeInBytes = sizeof(float) * outputChannelCount * inputChannelCount * heightOfChannels * widthOfChannels;
    hipMallocManaged(&(this->cuKernel.onDevice), this->cuKernel.sizeInBytes);

    checkCUDNN(hipdnnCreateConvolutionDescriptor(&this->cuConvolution.descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(this->cuConvolution.descriptor,
        /*pad_height=*/padding,
        /*pad_width=*/padding,
        /*vertical_stride=*/stride,
        /*horizontal_stride=*/stride,
        /*dilation_height=*/dilation,
        /*dilation_width=*/dilation,
        /*mode=*/HIPDNN_CROSS_CORRELATION,
        /*computeType=*/HIPDNN_DATA_FLOAT));
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(this->cuConvolution.descriptor,
        this->cuInput.descriptor,
        this->cuKernel.descriptor,
        &(this->cuOutput.batchSize),
        &(this->cuOutput.channelCount),
        &(this->cuOutput.height),
        &(this->cuOutput.width)));

    Tensor4D& cuOut = this->cuOutput;
    checkCUDNN(hipdnnCreateTensorDescriptor(&cuOut.descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(cuOut.descriptor,
        /*format=*/HIPDNN_TENSOR_NHWC,
        /*dataType=*/HIPDNN_DATA_FLOAT,
        /*batch_size=*/cuOut.batchSize,
        /*channels=*/cuOut.channelCount,
        /*image_height=*/cuOut.height,
        /*image_width=*/cuOut.width));
    cuOut.sizeInBytes = sizeof(float) * cuOut.batchSize * cuOut.channelCount * cuOut.height * cuOut.width;
    hipMallocManaged(&(cuOut.onDevice), cuOut.sizeInBytes);
    hipMemset(cuOut.onDevice, 0, cuOut.sizeInBytes);

    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(CuConvLayer::handle,
        this->cuInput.descriptor,
        this->cuKernel.descriptor,
        this->cuConvolution.descriptor,
        this->cuOutput.descriptor,
        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
        /*memoryLimitInBytes=*/0,
        &this->cuConvolution.algo));

    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(CuConvLayer::handle,
        this->cuInput.descriptor,
        this->cuKernel.descriptor,
        this->cuConvolution.descriptor,
        this->cuOutput.descriptor,
        this->cuConvolution.algo,
        &(this->cuWorkspace.sizeInBytes)));
    hipMallocManaged(&this->cuWorkspace.onDevice, this->cuWorkspace.sizeInBytes);

    this->activation = activation;
}

/// @brief Use if for creating hidden Conv input-layers
/// @param inputChannelCount 
/// @param outputChannelCount 
/// @param widthOfChannels 
/// @param heightOfChannels 
/// @param padding 
/// @param stride 
/// @param dilation 
/// @param prevLayer 
/// @param activation 
CuConvLayer::CuConvLayer(int inputChannelCount, int outputChannelCount, int widthOfChannels, int heightOfChannels,
    int padding, int stride, int dilation, 
    const CuConvLayer* prevLayer, Activation activation)
{
    this->cuInput = std::move(prevLayer->cuOutput);

    checkCUDNN(hipdnnCreateFilterDescriptor(&this->cuKernel.descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(this->cuKernel.descriptor,
        /*dataType=*/HIPDNN_DATA_FLOAT,
        /*format=*/HIPDNN_TENSOR_NCHW,
        /*out_channels=*/ (this->cuKernel.outChannels = outputChannelCount),
        /*in_channels=*/ (this->cuKernel.inChannels = inputChannelCount),
        /*kernel_height=*/ (this->cuKernel.heightOfChannel = heightOfChannels),
        /*kernel_width=*/ (this->cuKernel.widthOfChannel = widthOfChannels)));
    this->cuKernel.sizeInBytes = sizeof(float) * outputChannelCount * inputChannelCount * heightOfChannels * widthOfChannels;
    checkCUDA(hipMallocManaged(&(this->cuKernel.onDevice), this->cuKernel.sizeInBytes));

    checkCUDNN(hipdnnCreateConvolutionDescriptor(&this->cuConvolution.descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(this->cuConvolution.descriptor,
        /*pad_height=*/padding,
        /*pad_width=*/padding,
        /*vertical_stride=*/stride,
        /*horizontal_stride=*/stride,
        /*dilation_height=*/dilation,
        /*dilation_width=*/dilation,
        /*mode=*/HIPDNN_CROSS_CORRELATION,
        /*computeType=*/HIPDNN_DATA_FLOAT));
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(this->cuConvolution.descriptor,
        this->cuInput.descriptor,
        this->cuKernel.descriptor,
        &(this->cuOutput.batchSize),
        &(this->cuOutput.channelCount),
        &(this->cuOutput.height),
        &(this->cuOutput.width)));

    Tensor4D& cuOut = this->cuOutput;
    checkCUDNN(hipdnnCreateTensorDescriptor(&cuOut.descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(cuOut.descriptor,
        /*format=*/HIPDNN_TENSOR_NHWC,
        /*dataType=*/HIPDNN_DATA_FLOAT,
        /*batch_size=*/cuOut.batchSize,
        /*channels=*/cuOut.channelCount,
        /*image_height=*/cuOut.height,
        /*image_width=*/cuOut.width));
    cuOut.sizeInBytes = sizeof(float) * cuOut.batchSize * cuOut.channelCount * cuOut.height * cuOut.width;
    checkCUDA(hipMallocManaged(&(cuOut.onDevice), cuOut.sizeInBytes));
    checkCUDA(hipMemset(cuOut.onDevice, 0, cuOut.sizeInBytes));

    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(CuConvLayer::handle,
        this->cuInput.descriptor,
        this->cuKernel.descriptor,
        this->cuConvolution.descriptor,
        this->cuOutput.descriptor,
        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
        /*memoryLimitInBytes=*/0,
        &this->cuConvolution.algo));

    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(CuConvLayer::handle,
        this->cuInput.descriptor,
        this->cuKernel.descriptor,
        this->cuConvolution.descriptor,
        this->cuOutput.descriptor,
        this->cuConvolution.algo,
        &(this->cuWorkspace.sizeInBytes)));
    checkCUDA(hipMallocManaged(&this->cuWorkspace.onDevice, this->cuWorkspace.sizeInBytes));

    this->activation = activation;
}

CuConvLayer::~CuConvLayer()
{
}

/// @brief 
/// @param weights are in NCHW order
/// @param numberOfWeights 
/// @param bias it's not supported
/// @param numberOfBias it's not supported
void CuConvLayer::init(const float* weights, const int numberOfWeights, const float* bias, const int numberOfBias)
{
    CuKernel& cuKernel = this->cuKernel;
    if (numberOfWeights * sizeof(float) != cuKernel.sizeInBytes)
        throw "Number of weights received doesn't fit the expectation.";
    checkCUDA(hipMemcpy(cuKernel.onDevice, weights, cuKernel.sizeInBytes, hipMemcpyHostToDevice));
    hipDeviceSynchronize();
}

float* CuConvLayer::compute(const float* x)
{
    const float alpha = 1.0f, beta = 0.0f;

    if(this->hasInputLayer())
        checkCUDA(hipMemcpy(cuInput.onDevice, x, cuInput.sizeInBytes, hipMemcpyHostToDevice));

    checkCUDNN(hipdnnConvolutionForward(CuConvLayer::handle,
        &alpha,
        this->cuInput.descriptor,
        this->cuInput.onDevice,
        this->cuKernel.descriptor,
        this->cuKernel.onDevice,
        this->cuConvolution.descriptor,
        this->cuConvolution.algo,
        this->cuWorkspace.onDevice,
        this->cuWorkspace.sizeInBytes,
        &beta,
        this->cuOutput.descriptor,
        this->cuOutput.onDevice));
    hipDeviceSynchronize();

    #ifdef DEBUG
        std::cout << "\n=x=x=x=x=x=x=x=x=x=x=x=x=x=x=";
        int N = cuOutput.batchSize;
        int C = cuOutput.channelCount;
        int H = cuOutput.height;
        int W = cuOutput.width;
        float* data = new float[(long long)N * C * H * W];
        checkCUDA(hipMemcpy(data, cuOutput.onDevice, cuOutput.sizeInBytes, hipMemcpyDeviceToHost));
        int i = 0;
        for (int n = 0; n < N; n++)
        {
            std::cout << "\nBatch: " << N;
            for (int c = 0; c < C; c++)
            {
                std::cout << "\nChannel: " << c << " => \n";
                for (int h = 0; h < H; h++)
                {
                    for (int w = 0; w < W; w++)
                        std::cout << std::setw(6) << data[c + (w + (h + (n)*H) * W) * C] << "/" << data[i++];
                    std::cout << "\n";
                }
            }
        }
        delete(data);
    #endif // DEBUG
}

const Tensor4D& CuConvLayer::getOutputOnDevice() const
{
    return this->cuOutput;
}

std::vector<float>&& CuConvLayer::getOutput() const
{
    const Tensor4D& output = this->cuOutput;
    return std::vector<float>(output.onDevice, output.onDevice + output.sizeInBytes / sizeof(float));
}

