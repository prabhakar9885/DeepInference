#include "CudaEngine/Layers/CuConvLayer.cuh"

hipdnnHandle_t CuConvLayer::handle;

CuConvLayer::CuConvLayer(int inputChannelCount, int outputChannelCount, int widthOfChannels, int heightOfChannels, 
    int padding, int stride, int dilation,
    int inputImageBatchSize, int inputImageChannels, int inputImageHeight, int inputImageWidth, Activation activation)
{
    this->isInputLayer = true;
    checkCUDNN(hipdnnCreateTensorDescriptor(this->cuInput.descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(*this->cuInput.descriptor,
        /*format=*/HIPDNN_TENSOR_NHWC,
        /*dataType=*/HIPDNN_DATA_FLOAT,
        /*batch_size=*/inputImageBatchSize,
        /*channels=*/inputImageChannels,
        /*image_height=*/inputImageHeight,
        /*image_width=*/inputImageWidth));
    this->cuInput.sizeInBytes = sizeof(float) * inputImageBatchSize * inputImageChannels * inputImageHeight * inputImageWidth;
    hipMalloc(&(this->cuInput.onDevice), this->cuInput.sizeInBytes);

    checkCUDNN(hipdnnCreateFilterDescriptor(this->cuKernel.descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(*this->cuKernel.descriptor,
        /*dataType=*/HIPDNN_DATA_FLOAT,
        /*format=*/HIPDNN_TENSOR_NCHW,
        /*out_channels=*/outputChannelCount,
        /*in_channels=*/inputChannelCount,
        /*kernel_height=*/heightOfChannels,
        /*kernel_width=*/widthOfChannels));

    checkCUDNN(hipdnnCreateConvolutionDescriptor(this->cuConvolution.descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(*this->cuConvolution.descriptor,
        /*pad_height=*/padding,
        /*pad_width=*/padding,
        /*vertical_stride=*/stride,
        /*horizontal_stride=*/stride,
        /*dilation_height=*/dilation,
        /*dilation_width=*/dilation,
        /*mode=*/HIPDNN_CROSS_CORRELATION,
        /*computeType=*/HIPDNN_DATA_FLOAT));
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(*this->cuConvolution.descriptor,
        *this->cuInput.descriptor,
        *this->cuKernel.descriptor,
        &(this->cuOutput.batchSize),
        &(this->cuOutput.channelCount),
        &(this->cuOutput.height),
        &(this->cuOutput.width)));

    Tensor4D& cuOut = this->cuOutput;
    checkCUDNN(hipdnnCreateTensorDescriptor(cuOut.descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(*cuOut.descriptor,
        /*format=*/HIPDNN_TENSOR_NHWC,
        /*dataType=*/HIPDNN_DATA_FLOAT,
        /*batch_size=*/cuOut.batchSize,
        /*channels=*/cuOut.channelCount,
        /*image_height=*/cuOut.height,
        /*image_width=*/cuOut.width));
    cuOut.sizeInBytes = sizeof(float) * cuOut.batchSize * cuOut.channelCount * cuOut.height * cuOut.width;
    hipMalloc(&(cuOut.onDevice), cuOut.sizeInBytes);
    hipMemset(cuOut.onDevice, 0, cuOut.sizeInBytes);

    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(CuConvLayer::handle,
        *this->cuInput.descriptor,
        *this->cuKernel.descriptor,
        *this->cuConvolution.descriptor,
        *this->cuOutput.descriptor,
        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
        /*memoryLimitInBytes=*/0,
        this->cuConvolution.algo));

    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(CuConvLayer::handle,
        *this->cuInput.descriptor,
        *this->cuKernel.descriptor,
        *this->cuConvolution.descriptor,
        *this->cuOutput.descriptor,
        *this->cuConvolution.algo,
        &(this->cuWorkspace.sizeInBytes)));
    hipMalloc(&this->cuWorkspace.onDevice, this->cuWorkspace.sizeInBytes);

    this->activation = activation;
}

CuConvLayer::CuConvLayer(int inputChannelCount, int outputChannelCount, int widthOfChannels, int heightOfChannels,
    int padding, int stride, int dilation, 
    CuConvLayer* prevLayer, Activation activation)
{
    this->cuInput = std::move(prevLayer->cuOutput);

    checkCUDNN(hipdnnCreateFilterDescriptor(this->cuKernel.descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(*this->cuKernel.descriptor,
        /*dataType=*/HIPDNN_DATA_FLOAT,
        /*format=*/HIPDNN_TENSOR_NCHW,
        /*out_channels=*/outputChannelCount,
        /*in_channels=*/inputChannelCount,
        /*kernel_height=*/heightOfChannels,
        /*kernel_width=*/widthOfChannels));

    checkCUDNN(hipdnnCreateConvolutionDescriptor(this->cuConvolution.descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(*this->cuConvolution.descriptor,
        /*pad_height=*/padding,
        /*pad_width=*/padding,
        /*vertical_stride=*/stride,
        /*horizontal_stride=*/stride,
        /*dilation_height=*/dilation,
        /*dilation_width=*/dilation,
        /*mode=*/HIPDNN_CROSS_CORRELATION,
        /*computeType=*/HIPDNN_DATA_FLOAT));
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(*this->cuConvolution.descriptor,
        *this->cuInput.descriptor,
        *this->cuKernel.descriptor,
        &(this->cuOutput.batchSize),
        &(this->cuOutput.channelCount),
        &(this->cuOutput.height),
        &(this->cuOutput.width)));

    Tensor4D& cuOut = this->cuOutput;
    checkCUDNN(hipdnnCreateTensorDescriptor(cuOut.descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(*cuOut.descriptor,
        /*format=*/HIPDNN_TENSOR_NHWC,
        /*dataType=*/HIPDNN_DATA_FLOAT,
        /*batch_size=*/cuOut.batchSize,
        /*channels=*/cuOut.channelCount,
        /*image_height=*/cuOut.height,
        /*image_width=*/cuOut.width));
    cuOut.sizeInBytes = sizeof(float) * cuOut.batchSize * cuOut.channelCount * cuOut.height * cuOut.width;
    hipMalloc(&(cuOut.onDevice), cuOut.sizeInBytes);
    hipMemset(cuOut.onDevice, 0, cuOut.sizeInBytes);

    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(CuConvLayer::handle,
        *this->cuInput.descriptor,
        *this->cuKernel.descriptor,
        *this->cuConvolution.descriptor,
        *this->cuOutput.descriptor,
        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
        /*memoryLimitInBytes=*/0,
        this->cuConvolution.algo));

    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(CuConvLayer::handle,
        *this->cuInput.descriptor,
        *this->cuKernel.descriptor,
        *this->cuConvolution.descriptor,
        *this->cuOutput.descriptor,
        *this->cuConvolution.algo,
        &(this->cuWorkspace.sizeInBytes)));
    hipMalloc(&this->cuWorkspace.onDevice, this->cuWorkspace.sizeInBytes);

    this->activation = activation;
}

CuConvLayer::~CuConvLayer()
{
}
