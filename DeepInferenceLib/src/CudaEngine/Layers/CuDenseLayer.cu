
#include "CudaEngine/Layers/CuDenseLayer.cuh"
#define IDX2C(i,j,ld) (j*ld + i)

hipblasHandle_t CuDenseLayer::handle;

CuDenseLayer::CuDenseLayer(int sizeOfCurrentLayer, Activation activation) : sizeOfCurrentLayer{ sizeOfCurrentLayer }, activation{ activation }
{
    this->isInputLayer = true;
}

CuDenseLayer::CuDenseLayer(int sizeOfCurrentLayer, Activation activation, const CuLayer* prevLayer) : sizeOfCurrentLayer{ sizeOfCurrentLayer }, activation{ activation }
{
    this->prevLayer = prevLayer;
}

CuDenseLayer::~CuDenseLayer()
{
    std::cout << "\nCuDenseLayer->Destructor...";
    hipError_t status;
    if (aDevice && (status = hipFree(aDevice)) != hipSuccess)
    {
        std::cerr << "Failed to release device memory. Status code: " << status;
    }
    if (bDevice && (status = hipFree(bDevice)) != hipSuccess)
    {
        std::cerr << "Failed to release device memory. Status code: " << status;
    }
    hipblasStatus_t hipblasStatus_t;
    if (this->hasInputLayer() && (hipblasStatus_t = hipblasDestroy(CuDenseLayer::handle)) != HIPBLAS_STATUS_SUCCESS)
    {
        std::cerr << "Failed to destroy cublas-handle. Status code: " << hipblasStatus_t;
    }
}

void CuDenseLayer::setSizeOfInput(int sizeOfInput)
{
    this->sizeOfInput = sizeOfInput;
}

void CuDenseLayer::allocMemForLayer()
{
    if (this->hasInputLayer())
    {
        if (hipSuccess != hipMallocManaged((void**)&inputDevice, this->sizeOfInput * sizeof(float)))
            throw "Unable to allocate Input memory";
        hipblasCreate(&CuDenseLayer::handle);
    }
    this->aDeviceCount = this->sizeOfCurrentLayer * this->sizeOfInput;
    if (hipSuccess != hipMallocManaged((void**)&aDevice, this->aDeviceCount * sizeof(float)))
        throw "Unable to allocate memory";
    if (hipSuccess != hipMallocManaged((void**)&bDevice, this->sizeOfCurrentLayer * sizeof(float)))
        throw "Unable to allocate memory";
}

void CuDenseLayer::init(const float* weights, const int numberOfWeights, const float* bias, const int numberOfBias)
{
    if (this->sizeOfCurrentLayer != numberOfBias)
        throw "Size of bias and the number of nodes in the layer should match";
    if (this->aDeviceCount != numberOfWeights)
        throw " sizeOfPrevLayer * sizeOfCurrLayer != numberOfWeights";

    int wtIndx = 0;
    int sizeOfPreviousLayer = numberOfWeights / this->sizeOfCurrentLayer;
    for (int j = 0; j < sizeOfPreviousLayer; j++)
    {
        for (int i = 0; i < this->sizeOfCurrentLayer; i++)
        {
            this->aDevice[IDX2C(i, j, this->sizeOfCurrentLayer)] = weights[wtIndx];
            wtIndx++;
        }
    }
    for (int i = 0; i < this->sizeOfCurrentLayer; i++)
    {
        this->bDevice[i] = bias[i];
    }
}

float* CuDenseLayer::compute(const float* xDevice)
{
    float alpha = 1, beta = 1;
    if (this->hasInputLayer())
    {
        hipblasSetVector(this->sizeOfInput, sizeof(float), xDevice, 1, this->inputDevice, 1);
        xDevice = this->inputDevice;
    }
    int sizeOfPreviousLayer = this->aDeviceCount / this->sizeOfCurrentLayer;
    hipblasStatus_t status;
    status = hipblasSgemv(handle, HIPBLAS_OP_N, this->sizeOfCurrentLayer, sizeOfPreviousLayer, &alpha, this->aDevice, this->sizeOfCurrentLayer, xDevice, 1, &beta, this->bDevice, 1);
    if (status != HIPBLAS_STATUS_SUCCESS)
        throw "cuBLAS operation failure";
    if (this->activation != Activation::NONE)
        CuUtills::computeActivation(this->bDevice, this->sizeOfCurrentLayer, this->activation);
    hipDeviceSynchronize();
    return this->bDevice;
}

void CuDenseLayer::releaseMem()
{
    hipFree(this->aDevice);
    hipFree(this->bDevice);
}

std::vector<float>&& CuDenseLayer::getOutput() const
{
    return std::vector<float>(this->bDevice, this->bDevice + this->sizeOfCurrentLayer);
}

